#include "hip/hip_runtime.h"
// @file  mpn_cov_gpu.cu
// @brief MPN-COV implementation (GPU)
// @author Jiangtao Xie
// @author Peihua Li

/*
Copyright (C) 2017 Peihua Li and Jiangtao Xie

All rights reserved.
*/

#include "nncov_traceNorm_blas.hpp"
#include "../data.hpp"
#include <math.h>
#include <memory>
#include <cstdlib>
#include <algorithm>
#include <limits>
#include <cassert>
#include "blashelper_gpu.hpp"


#define CUDA_KERNEL_LOOP(i, n) \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
         i < (n); \
         i += blockDim.x * gridDim.x)   // We import this Macro function  from our Caffe Implementation 

inline int
GET_BLOCKS(const int N)
{
    return (N + VL_CUDA_NUM_THREADS - 1) / VL_CUDA_NUM_THREADS; // We import this function  from our Caffe Implementation 
}

template<typename T> __global__ void set_kernel(const ptrdiff_t n, const T alpha, T* y) 
{
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = alpha;
    }
}

template<typename T> void gpuMemset(const ptrdiff_t n, const T alpha, T* y)
{
    if(alpha == 0){
        hipMemset(y, 0, sizeof(T)*n);
    }
    set_kernel<T><<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(n , alpha, y);
}

template<typename T> __global__ void init_I_kernel(T* a,
												   T  alpha,
                                                   const ptrdiff_t n)
{
    CUDA_KERNEL_LOOP(index,n){
        a[index*(n+1)]  =  alpha;
    }
}




template<typename T> __global__ void traceNormBackward_kernel(T* a,
                                                              T const* alpha,
					                                          T* beta,
															  T const* sigma,
					                                          ptrdiff_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        a[i*(n+1)] = a[i*(n+1)] - beta[0]/(alpha[0] * alpha[0]) + sigma[0]/(2.0f * sqrt(alpha[0]));
    }
}

template<typename T>  __host__ void
traceNormBackward_gpu(T* a,
                      T const* alpha,
					  T* beta,
					  T const* sigma,
					  ptrdiff_t n)
{
    traceNormBackward_kernel<T>
		<<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(a,alpha,beta,sigma,n);
}

template<typename T> __global__ void copy_kernel(T* a,
                                                 T* b,
												 ptrdiff_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		a[i] = b[i];
	}
}

template<typename T> __global__ void scale_kernel(T* a,
												  T const* b,
                                                  T const* alpha,
												  ptrdiff_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		a[i] = (1.0f / alpha[0]) * b[i];
	}
}
template<typename T> __global__ void scale2_kernel(T* a,
												   T const* b,
                                                   T const* alpha,
												   ptrdiff_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		a[i] = sqrt(alpha[0]) * b[i];
	}
}
template<typename T> __global__ void mul_kernel(T* a,
                                                T const* alpha,
												ptrdiff_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		a[i] = a[i] / ( 2.0f * alpha[0]);
	}
}
namespace vl { namespace impl {
    template<typename T,vl::DataType dataType>
    struct cov_traceNorm<vl::VLDT_GPU,T,dataType>
    {
        static vl::ErrorCode
            forward(Context& context,
                    T* output,
                    T const* data,
					T* aux_T,
                    size_t height, size_t width, size_t depth, size_t num)
        {
            vl::ErrorCode error;
            ptrdiff_t m = height,n = width,L = num,d,i;
			ptrdiff_t dataOffset;
			ptrdiff_t aux_TOffset;
			ptrdiff_t outputOffset;
			unsigned int workspaceSize =  (unsigned int)(n*n);
			T* workspace = (T*)context.getWorkspace(vl::VLDT_GPU , workspaceSize*sizeof(T));
			T* I1        = workspace;
			gpuMemset(n*n, T(0), I1);
			init_I_kernel<T><<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(I1,(T)1,n);
			for(d = 0;d < L; d++){ // Trace Norm
				aux_TOffset  = d;
				outputOffset = d*n*n;
				dataOffset   = d*n*n;
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::dot(context,
					                                                n*n,
																	data + dataOffset,ptrdiff_t(1),
															        I1,ptrdiff_t(1),
															        aux_T + aux_TOffset);
				if(error != vl::VLE_Success) {goto done ;}
			}
			for(d = 0;d < L; d++){
				aux_TOffset  = d;
				outputOffset = d*n*n;
				dataOffset   = d*n*n;
				scale_kernel<T><<<GET_BLOCKS(n*n),VL_CUDA_NUM_THREADS>>>(output + outputOffset,data + dataOffset,aux_T + aux_TOffset,n*n);
			}
            
        done:
            return context.passError(error, __func__);
        }
        static vl::ErrorCode
            backward(Context& context,
                     T* derData,
                     T const* data,
                     T const* derOutput,
					 T const* derOutput_aux,
					 T const* aux_T,
                     size_t height, size_t width, size_t depth, size_t num)
        {
            vl::ErrorCode error;
			ptrdiff_t m = height,n = width,L = num,d;
			ptrdiff_t derDataOffset,aux_TOffset,derOutputOffset;
			ptrdiff_t dataOffset,derOutput_auxOffset,P_dot_dLdPOffset;
			unsigned int workspaceSize =  (unsigned int)(L);
			T* workspace = (T*)context.getWorkspace(vl::VLDT_GPU , workspaceSize*sizeof(T));
			T* P_dot_dLdP = workspace;
			/*debug
			mxArray *x;
            mxClassID classID = mxSINGLE_CLASS ;
            mwSize dim[2];
            dim[0] = n;dim[1] = n;
            x  = mxCreateNumericArray(2,dim,classID,mxREAL);
		 	T *test = (T*)mxGetData(x);
			memcpy(test,YZ,sizeof(T)*n*n);
            mexCallMATLAB(0,NULL,1,&x,"Watch");
			*/
			for(d = 0;d < L;d++){
				dataOffset    = d*m*n;
				derDataOffset = d*m*n;
				derOutputOffset = d*m*n;
				derOutput_auxOffset = d;
				aux_TOffset   = d;
				P_dot_dLdPOffset = d;
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::dot(context,
					                                                n*n,
															        data + dataOffset,ptrdiff_t(1),
																	derOutput + derOutputOffset,ptrdiff_t(1),
															        P_dot_dLdP + P_dot_dLdPOffset);
				if(error != vl::VLE_Success) {goto done ;}
			}
			for(d = 0;d < L; d++){
				dataOffset    = d*m*n;
				derDataOffset = d*m*n;
				derOutputOffset = d*m*n;
				derOutput_auxOffset = d;
				aux_TOffset   = d;
				P_dot_dLdPOffset = d;
				scale_kernel<T><<<GET_BLOCKS(n*n),VL_CUDA_NUM_THREADS>>>(derData + derDataOffset,derOutput + derOutputOffset,aux_T + aux_TOffset,n*n);
				traceNormBackward_gpu(derData + derDataOffset,aux_T + aux_TOffset,P_dot_dLdP + P_dot_dLdPOffset,derOutput_aux + derOutput_auxOffset, n);
			}

            done:
            return context.passError(error, __func__);
           
        }
		static vl::ErrorCode
			forward_aux(Context& context,
			            T* output,
						T const* data,
						T* aux_T,
						size_t height, size_t width, size_t depth, size_t num)
		{
			vl::ErrorCode error;
			ptrdiff_t length = depth,L = num,d;
			ptrdiff_t outputOffset,aux_TOffset,dataOffset;
			for(d = 0; d < L; d++){
				outputOffset = d*length;
				dataOffset = d*length;
				aux_TOffset  = d;
				/*error = vl::impl::blas<vl::VLDT_CPU, dataType>::scal(context,
					                                                 length,
																	 alpha,
																	 output + OutputOffset,ptrdiff_t(1));
				if(error != vl::VLE_Success) {goto done ;}*/
				scale2_kernel<T><<<GET_BLOCKS(length),VL_CUDA_NUM_THREADS>>>(output + outputOffset,data + dataOffset,aux_T + aux_TOffset,length);
				error = vl::VLE_Success;
			}
			done:
            return context.passError(error, __func__);
		}
		static vl::ErrorCode
			backward_aux(Context& context,
			             T* derData,
						 T* derData_aux,
						 T const* data,
						 T const* derOutput,
						 T const* aux_T,
						 size_t height, size_t width, size_t depth, size_t num)
		{
			vl::ErrorCode error;
			ptrdiff_t length = depth,L = num,d;
			ptrdiff_t dataOffset,aux_TOffset;
			ptrdiff_t derDataOffset,derData_auxOffset;
			ptrdiff_t derOutputOffset;
			for(d = 0; d < L; d++){
				derDataOffset   = d*length;
				derOutputOffset = d*length;
				derData_auxOffset = d;
				dataOffset = d*length;
				aux_TOffset= d;
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::dot(context,
					                                                length,
															        data + dataOffset,ptrdiff_t(1),
																	derOutput + derOutputOffset,ptrdiff_t(1),
																	derData_aux + derData_auxOffset);
				if(error != vl::VLE_Success) {goto done ;}
				
			}
			for(d = 0; d < L; d++){
				derDataOffset   = d*length;
				derOutputOffset = d*length;
				derData_auxOffset = d;
				dataOffset = d*length;
				aux_TOffset= d;
				scale2_kernel<T><<<GET_BLOCKS(length),VL_CUDA_NUM_THREADS>>>(derData + derDataOffset,derOutput + derOutputOffset,aux_T + aux_TOffset,length);
			}
			done:
            return context.passError(error, __func__);
		}


    };
} }
template struct vl::impl::cov_traceNorm<vl::VLDT_GPU, float,vl::VLDT_Float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::cov_traceNorm<vl::VLDT_GPU, double, vl::VLDT_Double> ;
#endif
