#include "hip/hip_runtime.h"
// @file  mpn_cov_gpu.cu
// @brief MPN-COV implementation (GPU)
// @author Jiangtao Xie
// @author Peihua Li

/*
Copyright (C) 2017 Peihua Li and Jiangtao Xie

All rights reserved.
*/

#include "nncov_sqrtm_blas.hpp"
#include "../data.hpp"
#include <math.h>
#include <memory>
#include <cstdlib>
#include <algorithm>
#include <limits>
#include <cassert>
#include "blashelper_gpu.hpp"


#define CUDA_KERNEL_LOOP(i, n) \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
         i < (n); \
         i += blockDim.x * gridDim.x)   // We import this Macro function  from our Caffe Implementation 

inline int
GET_BLOCKS(const int N)
{
    return (N + VL_CUDA_NUM_THREADS - 1) / VL_CUDA_NUM_THREADS; // We import this function  from our Caffe Implementation 
}

template<typename T> __global__ void set_kernel(const ptrdiff_t n, const T alpha, T* y) 
{
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = alpha;
    }
}

template<typename T> void gpuMemset(const ptrdiff_t n, const T alpha, T* y)
{
    if(alpha == 0){
        hipMemset(y, 0, sizeof(T)*n);
    }
    set_kernel<T><<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(n , alpha, y);
}
template<typename T> __global__ void init_II_kernel(T* a,
                                                   T aux_value,
                                                   const ptrdiff_t n)
{
    CUDA_KERNEL_LOOP(index,n){
        a[index*(n+1)]  =  a[index*(n+1)] + aux_value;
    }
}
template<typename T> __global__ void init_I_kernel(T* a,
												   T  alpha,
                                                   const ptrdiff_t n)
{
    CUDA_KERNEL_LOOP(index,n){
        a[index*(n+1)]  =  alpha;
    }
}

template<typename T> __global__ void getOutput_kernel(T* a,
                                                      T* b,
													  T coef,
                                                      ptrdiff_t n)
{
    int lda = n,offset,idx = 0;
    CUDA_KERNEL_LOOP(i,n){
        offset = i;
        for(int j = 0;j < offset + 1;j ++) {
            idx = i * (i + 1) / 2.0f + j;
            a[idx] = coef*b[i * lda + j];
        }
    }
}
template<typename T>  __host__ void
getOutput_gpu(T* output,
              T* result,
			  T coef,
              ptrdiff_t n)
{
    getOutput_kernel<T><<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(output,result,coef,n);
}
template<typename T> __global__ void getdLdCfromderOutput_kernel(T* a,
                                                                 T const* b,
																 T coef,
                                                                 ptrdiff_t n)
{
    int lda = n,offset,idx = 0;
    CUDA_KERNEL_LOOP(i,n){
        offset = i;
        for(int j = 0;j < offset + 1;j ++){
            idx = i * (i + 1) / 2.0f + j;
            a[i * lda + j] = coef*b[idx];
        }
    }
}
template<typename T>  __host__ void
getdLdCfromderOutput_gpu(T* dLdC,
                         T const* derOutput,
						 T coef,
                         ptrdiff_t n)
{
    getdLdCfromderOutput_kernel<T><<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(dLdC,derOutput,coef,n);
}

template<typename T> __global__ void symmetric_kernel(T* a,
                                                      int n)
{
    int lda = n,offset;
    CUDA_KERNEL_LOOP(i,n){
        offset = i;
        for(int j = offset;j < n;j ++) {
            a[i * lda + j] = (a[i * lda + j] + a[j * lda + i]) / 2.0f;
            a[j * lda + i] = a[i * lda + j];
        }
    }
}

template<typename T>  __host__ void
symmetric_gpu(T* a,
              ptrdiff_t n)
{
    symmetric_kernel<T>
        <<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(a,n);
}

template<typename T> __global__ void matrixAdd_kernel(T* x,
                                                      T* y,
		                                              T* z,
		                                              T alpha,
		                                              T beta,
		                                              T sigma,
		                                              ptrdiff_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        x[i] = alpha * x[i] + beta * y[i] + sigma * z[i];
    }
}
template<typename T> __host__ void
matrixAdd_gpu(T* x,
              T* y,
		      T* z,
		      T alpha,
		      T beta,
		      T sigma,
		      ptrdiff_t n)
{
    matrixAdd_kernel<T>
        <<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(x,y,z,alpha,beta,sigma,n);
}
template<typename T> __global__ void copy_kernel(T* a,
                                                 T const* b,
												 ptrdiff_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n){
		a[i] = b[i];
	}
}
namespace vl { namespace impl {
    template<typename T,vl::DataType dataType>
    struct cov_sqrtm<vl::VLDT_GPU,T,dataType>
    {
        static vl::ErrorCode
            forward(Context& context,
                    T* output,
                    T const* data,
                    T* aux_Y,
                    T* aux_Z,
                    size_t height, size_t width, size_t depth, size_t num,
					int coef,
                    int iterNum)
        {
           vl::ErrorCode error;
            ptrdiff_t m = height,n = width,L = num,d,i;
			ptrdiff_t dataOffset,ypOffset;
			ptrdiff_t aux_YOffset,aux_ZOffset,aux_TOffset;
			ptrdiff_t aux_YOffset_1,aux_ZOffset_1,outputOffset;
			unsigned int workspaceSize =  (unsigned int)(n*n*2);
			T* workspace = (T*)context.getWorkspace(vl::VLDT_GPU , workspaceSize*sizeof(T));
			T* I3        = workspace;
			T* result    = I3 + n*n;

			T* ZY        = NULL;
			gpuMemset(n*n, T(0), I3);
			init_I_kernel<T><<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(I3,(T)3,n);
			for(d = 0;d < L; d++){
				ZY    = aux_Z + n*n*iterNum*d + n*n; // Z1
				dataOffset = d*n*n; 
				aux_YOffset = n*n*iterNum*d + n*n; //Y1
				ypOffset = d*(n*n*iterNum);
				outputOffset = d*n*(n+1)/2;
				copy_kernel<T><<<GET_BLOCKS(n*n),VL_CUDA_NUM_THREADS>>>(ZY,data + dataOffset,n*n);
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::axpy(context,
					                                                 n*n,(T)(-1),
															         I3,ptrdiff_t(1),
															         ZY,ptrdiff_t(1));
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
					                                                 'n','n',
															         n,n,n,
															         (T)(-0.5),data + dataOffset,n,
															         ZY,n,
																	 (T)0,aux_Y + aux_YOffset,n);
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::scal(context,
					                                                 n*n,
					                                                (T)(-0.5),
															         ZY,ptrdiff_t(1));
				if(error != vl::VLE_Success) {goto done ;}
				for(i = 2;i < iterNum;i++){
					ZY = aux_Z + n*n*iterNum*d;
					aux_YOffset = n*n*iterNum*d + i*n*n;
					aux_ZOffset = n*n*iterNum*d + i*n*n;
					aux_YOffset_1 = n*n*iterNum*d + (i-1)*n*n;
					aux_ZOffset_1 = n*n*iterNum*d + (i-1)*n*n;
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																          n,n,n,
																         (T)1,aux_Z + aux_ZOffset_1,n,
																          aux_Y + aux_YOffset_1,n,
																         (T)0,ZY,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::axpy(context,
						                                                 n*n,(T)(-1),
																         I3,ptrdiff_t(1),
															             ZY,ptrdiff_t(1));
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																          n,n,n,
																         (T)(-0.5),aux_Y + aux_YOffset_1,n,
																          ZY,n,
																         (T)0,aux_Y + aux_YOffset,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																        (T)(-0.5),ZY,n,
																         aux_Z + aux_ZOffset_1,n,
																         (T)0,aux_Z + aux_ZOffset,n);
					if(error != vl::VLE_Success) {goto done ;}
				}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                             'n','n',
														              n,n,n,
														             (T)1,aux_Z + aux_ZOffset,n,
															         aux_Y + aux_YOffset,n,
															         (T)0,ZY,n);
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::axpy(context,
						                                             n*n,(T)(-1),
														             I3,ptrdiff_t(1),
															         ZY,ptrdiff_t(1));
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                             'n','n',
														             n,n,n,
														             (T)(-0.5),aux_Y + aux_YOffset,n,
														             ZY,n,
														             (T)0,result,n);
				if(error != vl::VLE_Success) {goto done ;}
				getOutput_gpu(output + outputOffset,result, (T)coef, n);
			}
            
        done:
            return context.passError(error, __func__);
        }
        static vl::ErrorCode
            backward(Context& context,
                     T* derData,
                     T const* data,
                     T const* derOutput,
                     T const* aux_Y,
                     T const* aux_Z,
                     size_t height, size_t width, size_t depth, size_t num,
					 int coef,
                     int iterNum)
        {
            vl::ErrorCode error;
			ptrdiff_t m = height,n = width,L = num,d,i;
			ptrdiff_t derOutputOffset,dLdCOffset,dataOffset;
			ptrdiff_t aux_YOffset,aux_ZOffset,aux_TOffset;
			ptrdiff_t aux_YOffset_1,aux_ZOffset_1;
			ptrdiff_t dLdYOffset,dLdZOffset;
			ptrdiff_t dLdYOffset_1,dLdZOffset_1;
			ptrdiff_t derDataOffset;
			unsigned int workspaceSize =  (unsigned int)(n*n*(iterNum)*2 + n*n*L + n*n*7);
			T* workspace = (T*)context.getWorkspace(vl::VLDT_GPU , workspaceSize*sizeof(T));
			T* dLdY      = workspace;
			T* dLdZ      = workspace + n*n*(iterNum);
			T* I3        = dLdZ + n*n*(iterNum);
			T* dLdC      = I3 + n*n;
			T* iterMemA  = dLdC + n*n*L;
			T* iterMemB  = iterMemA + n*n;
			T* iterMemC  = iterMemB + n*n;
			T* ZY        = iterMemC + n*n;
			T* YZ        = ZY + n*n;
			T* ZY_dLdY   = NULL;T* dLdZ_ZY   = NULL;
			T* Z_dLdZ    = NULL;T* Y_dLdY    = NULL;
			T* Z_dLdZ_Z  = NULL;T* Y_dLdY_Y  = NULL;
			T* dLdX      = NULL;T* dLdP      = NULL;
			T  const* P  = NULL;
			T  P_dot_dLdP;
			gpuMemset(n*n, T(0), I3);
			gpuMemset(n*n*L, T(0), dLdC);
			init_I_kernel<T><<<GET_BLOCKS(n),VL_CUDA_NUM_THREADS>>>(I3,(T)3,n);
			/*debug
			mxArray *x;
            mxClassID classID = mxSINGLE_CLASS ;
            mwSize dim[2];
            dim[0] = n;dim[1] = n;
            x  = mxCreateNumericArray(2,dim,classID,mxREAL);
		 	T *test = (T*)mxGetData(x);
			memcpy(test,YZ,sizeof(T)*n*n);
            mexCallMATLAB(0,NULL,1,&x,"Watch");
			*/
			for(d = 0;d < L;d++){
				derOutputOffset = d*n*(n+1)/2;
				dLdCOffset      = d*n*n;
				getdLdCfromderOutput_gpu(dLdC + dLdCOffset, derOutput + derOutputOffset,(T)coef, n);
				//symmetric_cpu(dLdC + derOutputOffset,n);
			}
			for(d = 0;d < L;d++){
				derDataOffset = d*m*n;
				aux_TOffset = d;
				aux_YOffset = n*n*iterNum*d + n*n*(iterNum - 1);
				aux_ZOffset = n*n*iterNum*d + n*n*(iterNum - 1);
				dLdYOffset  = n*n*(iterNum - 1);
				dLdZOffset  = n*n*(iterNum - 1);
				dataOffset  = d*m*n;
				dLdCOffset  = d*n*n;
				ZY_dLdY     = iterMemA;
				Y_dLdY      = iterMemC;
				P           = data + dataOffset;
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
					                                                 'n','n',
															          n,n,n,
															         (T)1,aux_Y + aux_YOffset,n,
															          aux_Z + aux_ZOffset,n,
															         (T)0,YZ,n);
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::axpy(context,
					                                                 n*n,(T)(-1),
															         I3,ptrdiff_t(1),
															         YZ,ptrdiff_t(1));
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
					                                                 'n','n',
															         n,n,n,
															         (T)1,aux_Z + aux_ZOffset,n,
															         aux_Y + aux_YOffset,n,
															         (T)0,ZY,n);
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
					                                                 'n','n',
															         n,n,n,
															         (T)(-0.5),ZY,n,
															         dLdC + dLdCOffset,n,
															         (T)0,ZY_dLdY,n);
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
					                                                 'n','n',
															         n,n,n,
															         (T)(-0.5),dLdC + dLdCOffset,n,
															          YZ,n,
															         (T)0,dLdY + dLdYOffset,n);
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::axpy(context,
					                                                 n*n,(T)1,
															         ZY_dLdY,ptrdiff_t(1),
															         dLdY + dLdYOffset,ptrdiff_t(1));
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
					                                                 'n','n',
																	 n,n,n,
															         (T)1,aux_Y + aux_YOffset,n,
															         dLdC + dLdCOffset,n,
															         (T)0,Y_dLdY,n);
				if(error != vl::VLE_Success) {goto done ;}
				error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
					                                                 'n','n',
															          n,n,n,
															          (T)(-0.5),Y_dLdY,n,
															          aux_Y + aux_YOffset,n,
															          (T)0,dLdZ + dLdZOffset,n);
				if(error != vl::VLE_Success) {goto done ;}
				for(i = iterNum - 1;i > 1;i--){
					dLdYOffset = n*n*(i);
					dLdZOffset = n*n*(i);
					dLdYOffset_1 = n*n*(i-1);
					dLdZOffset_1 = n*n*(i-1);
					aux_YOffset_1  = n*n*iterNum*d + n*n*(i-1);
					aux_ZOffset_1  = n*n*iterNum*d + n*n*(i-1);
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																          n,n,n,
																          (T)1,aux_Y + aux_YOffset_1,n,
																          aux_Z + aux_ZOffset_1,n,
																          (T)0,YZ,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::axpy(context,
					                                                     n*n,(T)(-1),
															             I3,ptrdiff_t(1),
															             YZ,ptrdiff_t(1));
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)1,aux_Z+ aux_ZOffset_1,n,
																         aux_Y + aux_YOffset_1,n,
																         (T)0,ZY,n);
					if(error != vl::VLE_Success) {goto done ;}
					Z_dLdZ = iterMemC;Z_dLdZ_Z = iterMemB;ZY_dLdY = iterMemA;
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)1,aux_Z + aux_ZOffset_1,n,
																         dLdZ + dLdZOffset,n,
																         (T)0,Z_dLdZ,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)(-0.5),Z_dLdZ,n,
																         aux_Z + aux_ZOffset_1,n,
																         (T)0,Z_dLdZ_Z,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)(-0.5),ZY,n,
																         dLdY + dLdYOffset,n,
																         (T)0,ZY_dLdY,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)(-0.5),dLdY + dLdYOffset,n,
																         YZ,n,
																         (T)0,dLdY + dLdYOffset_1,n);
					if(error != vl::VLE_Success) {goto done ;}
					matrixAdd_gpu(dLdY + dLdYOffset_1,Z_dLdZ_Z,ZY_dLdY,T(1),T(1),T(1),n*n);
					Y_dLdY = iterMemC;Y_dLdY_Y = iterMemB;dLdZ_ZY = iterMemA;
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)1,aux_Y + aux_YOffset_1,n,
																         dLdY + dLdYOffset,n,
																         (T)0,Y_dLdY,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)(-0.5),Y_dLdY,n,
																         aux_Y + aux_YOffset_1,n,
																         (T)0,Y_dLdY_Y,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
																         (T)(-0.5),dLdZ + dLdZOffset,n,
																         ZY,n,
																         (T)0,dLdZ_ZY,n);
					if(error != vl::VLE_Success) {goto done ;}
					error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                                 'n','n',
																         n,n,n,
											 					         (T)(-0.5),YZ,n,
																         dLdZ + dLdZOffset,n,
																         (T)0,dLdZ + dLdZOffset_1,n);
					if(error != vl::VLE_Success) {goto done ;}
					matrixAdd_gpu(dLdZ + dLdZOffset_1,Y_dLdY_Y,dLdZ_ZY,T(1),T(1),T(1),n*n);
				}
				dLdP  = derData + derDataOffset;
                error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                             'n','n',
														              n,n,n,
															         (T)(-0.5),dLdY + dLdYOffset_1,n,
															          P,n,
															         (T)0,dLdP,n);
				if(error != vl::VLE_Success) {goto done ;}
                error = vl::impl::blas<vl::VLDT_GPU, dataType>::gemm(context,
						                                             'n','n',
														             n,n,n,
															         (T)(-0.5),P,n,
															         dLdY + dLdYOffset_1,n,
															         (T)(1),dLdP,n);
				if(error != vl::VLE_Success) {goto done ;}
				matrixAdd_gpu(dLdP,dLdY + dLdYOffset_1,dLdZ + dLdZOffset_1,T(1),T(1.5),T(-0.5),n*n);

			}

            done:
            return context.passError(error, __func__);
           
        }


    };
} }
template struct vl::impl::cov_sqrtm<vl::VLDT_GPU, float,vl::VLDT_Float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::cov_sqrtm<vl::VLDT_GPU, double, vl::VLDT_Double> ;
#endif
