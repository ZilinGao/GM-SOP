// @file  nncov_traceNorm.cu
// @brief MPN-COV block
// @author Jiangtao Xie
// @author Peihua Li

/*
Copyright (C) 2017 Peihua Li and Jiangtao Xie

All rights reserved.
*/


#include "nncov_traceNorm.hpp"
#include "impl/nncov_traceNorm_blas.hpp"
#if ENABLE_CUDNN
#include "impl/nncov_traceNorm_blas.hpp" // cudnn -> blas
#endif
#include <assert.h>

#ifdef ENABLE_GPU
#include "datacu.hpp"
#endif
using namespace vl ;

#define DISPATCH(deviceType, type, dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::forward \
(context, \
(type*)output.getMemory(), \
(type const*)data.getMemory(), \
(type*)aux_T.getMemory(), \
data.getHeight(),data.getWidth(),data.getDepth(),data.getSize()) ;

#define DISPATCH2(deviceType) \
switch (dataType) { \
case VLDT_Float : DISPATCH(deviceType, float, VLDT_Float) ; break ; \
    IF_DOUBLE(case VLDT_Double : DISPATCH(deviceType, double, VLDT_Double) ; break ;)\
default: assert(false) ; return VLE_Unknown ; \
}

#define DISPATCHCUDNN(dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::forward \
(context, \
(type*)output.getMemory(), \
(type const*)data.getMemory(), \
(type*)aux_T.getMemory(), \
data.getHeight(),data.getWidth(),data.getDepth(),data.getSize()) ;

#define DISPATCHCUDNN2() \
switch (dataType) { \
case VLDT_Float : DISPATCHCUDNN(VLDT_Float) ; break ; \
IF_DOUBLE(case VLDT_Double : DISPATCHCUDNN(VLDT_Double) ; break ;) \
default: assert(false) ; return VLE_Unknown ; \
}

vl::ErrorCode
vl::nncov_traceNorm_forward(vl::Context& context,
                            vl::Tensor output,
                            vl::Tensor data,
					        vl::Tensor aux_T)
{
  vl::ErrorCode error = VLE_Success ;
  vl::DataType dataType = output.getDataType() ;

  switch (output.getDeviceType()) {
    default:
      assert(false) ;
      error = vl::VLE_Unknown ;
      break ;

    case vl::VLDT_CPU:
      DISPATCH2(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
    case vl::VLDT_GPU:
#if ENABLE_CUDNN
//      if (context.getCudaHelper().getCudnnEnabled()) {
//        DISPATCHCUDNN2() ;
//        if (error == vl::VLE_Success) { return error ; }
//        if (error != vl::VLE_Unsupported) { goto done ; }
        /* this case was not supported by CUDNN -- fallback */
//      }
#endif
      DISPATCH2(vl::VLDT_GPU) ;
      break ;
#endif
  }
#if ENABLE_CUDNN
//done:
#endif
  return context.passError(error,__func__) ;
}


#undef DISPATCH
#define DISPATCH(deviceType, type, dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::backward \
(context, \
 (type*)derData.getMemory(), \
 (type const*)data.getMemory(),    \
 (type const*)derOutput.getMemory(), \
 (type const*)derOutput_aux.getMemory(), \
 (type const*)aux_T.getMemory(), \
 derData.getHeight(),derData.getWidth(),derData.getDepth(),derData.getSize()) ;

#undef DISPATCHCUDNN
#define DISPATCHCUDNN(dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::backward \
(context, \
 (type*)derData.getMemory(), \
 (type const*)data.getMemory(),    \
 (type const*)derOutput.getMemory(), \
 (type const*)derOutput_aux.getMemory(), \
 (type const*)aux_T.getMemory(), \
 derData.getHeight(),derData.getWidth(),derData.getDepth(),derData.getSize()) ;

vl::ErrorCode
vl::nncov_traceNorm_backward(vl::Context& context,
                             vl::Tensor derData,
                             vl::Tensor data,
                             vl::Tensor derOutput,
					         vl::Tensor derOutput_aux,
					         vl::Tensor aux_T)
{
  vl::ErrorCode error = vl::VLE_Success ;
  vl::DataType dataType = derOutput.getDataType() ;

  switch (derOutput.getDeviceType()) {
    default:
      assert(false) ;
      error = vl::VLE_Unknown ;
      break ;

    case vl::VLDT_CPU:
      DISPATCH2(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
    case vl::VLDT_GPU:

#if ENABLE_CUDNN
//     if (context.getCudaHelper().getCudnnEnabled()) {
//        DISPATCHCUDNN2() ;
//        if (error == vl::VLE_Success) { return error ; }
//        if (error != vl::VLE_Unsupported) { goto done ; }
        /* this case was not supported by CUDNN -- fallback */
//      }
#endif
      DISPATCH2(vl::VLDT_GPU) ;
      break ;
#endif
  }
#if ENABLE_CUDNN
//done:
#endif
  return context.passError(error,__func__) ;
}


/* ---------------------------------------------------------------- */
/*                                                          auxMode */
/* ---------------------------------------------------------------- */


#undef DISPATCH
#define DISPATCH(deviceType, type, dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::forward_aux \
(context, \
(type*)output.getMemory(), \
(type const*)data.getMemory(), \
(type*)aux_T.getMemory(), \
data.getHeight(),data.getWidth(),data.getDepth(),data.getSize()) ;

#undef DISPATCHCUDNN
#define DISPATCHCUDNN(dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::forward_aux \
(context, \
(type*)output.getMemory(), \
(type const*)data.getMemory(), \
(type*)aux_T.getMemory(), \
data.getHeight(),data.getWidth(),data.getDepth(),data.getSize()) ;


vl::ErrorCode
vl::nncov_traceNorm_aux_forward(vl::Context& context,   //
                            vl::Tensor output,
						    vl::Tensor data,
							vl::Tensor aux_T)
{
  vl::ErrorCode error = VLE_Success ;
  vl::DataType dataType = output.getDataType() ;

  switch (output.getDeviceType()) {
    default:
      assert(false) ;
      error = vl::VLE_Unknown ;
      break ;

    case vl::VLDT_CPU:
      DISPATCH2(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
    case vl::VLDT_GPU:
#if ENABLE_CUDNN
//      if (context.getCudaHelper().getCudnnEnabled()) {
//        DISPATCHCUDNN2() ;
//        if (error == vl::VLE_Success) { return error ; }
//        if (error != vl::VLE_Unsupported) { goto done ; }
        /* this case was not supported by CUDNN -- fallback */
//      }
#endif
      DISPATCH2(vl::VLDT_GPU) ;
      break ;
#endif
  }
#if ENABLE_CUDNN
//done:
#endif
  return context.passError(error,__func__) ;
}


#undef DISPATCH
#define DISPATCH(deviceType, type, dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::backward_aux \
(context, \
 (type*)derData.getMemory(), \
 (type*)derData_aux.getMemory(),    \
 (type const*)data.getMemory(), \
 (type const*)derOutput.getMemory(), \
 (type const*)aux_T.getMemory(), \
 derData.getHeight(),derData.getWidth(),derData.getDepth(),derData.getSize()) ;

#undef DISPATCHCUDNN
#define DISPATCHCUDNN(dataType) \
error = vl::impl::cov_traceNorm<deviceType, type, dataType>::backward_aux \
(context, \
 (type*)derData.getMemory(), \
 (type*)derData_aux.getMemory(),    \
 (type*)data.getMemory(), \
 (type const*)derOutput.getMemory(), \
 derData.getHeight(),derData.getWidth(),derData.getDepth(),derData.getSize()) ;

vl::ErrorCode
vl::nncov_traceNorm_aux_backward(vl::Context& context,
                               vl::Tensor derData,
							   vl::Tensor derData_aux,
                               vl::Tensor data,
                               vl::Tensor derOutput,
							   vl::Tensor aux_T)
{
  vl::ErrorCode error = vl::VLE_Success ;
  vl::DataType dataType = derOutput.getDataType() ;

  switch (derOutput.getDeviceType()) {
    default:
      assert(false) ;
      error = vl::VLE_Unknown ;
      break ;

    case vl::VLDT_CPU:
      DISPATCH2(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
    case vl::VLDT_GPU:

#if ENABLE_CUDNN
//     if (context.getCudaHelper().getCudnnEnabled()) {
//        DISPATCHCUDNN2() ;
//        if (error == vl::VLE_Success) { return error ; }
//        if (error != vl::VLE_Unsupported) { goto done ; }
        /* this case was not supported by CUDNN -- fallback */
//      }
#endif
      DISPATCH2(vl::VLDT_GPU) ;
      break ;
#endif
  }
#if ENABLE_CUDNN
//done:
#endif
  return context.passError(error,__func__) ;
}