// @file  nncov_pool.cu
// @brief MPN-COV block
// @author Jiangtao Xie
// @author Peihua Li

/*
Copyright (C) 2017 Peihua Li and Jiangtao Xie

All rights reserved.
*/


#include "nncov_pool.hpp"
#include "impl/nncov_pool_blas.hpp"
#if ENABLE_CUDNN
#include "impl/nncov_pool_blas.hpp" // cudnn -> blas
#endif
#include <assert.h>

#ifdef ENABLE_GPU
#include "datacu.hpp"
#endif
using namespace vl ;

#define DISPATCH(deviceType, type, dataType) \
error = vl::impl::cov_pool<deviceType, type, dataType>::forward \
(context, \
(type*)output.getMemory(), \
(type const*)data.getMemory(), \
data.getHeight(),data.getWidth(),data.getDepth(),data.getSize()) ;

#define DISPATCH2(deviceType) \
switch (dataType) { \
case VLDT_Float : DISPATCH(deviceType, float, VLDT_Float) ; break ; \
    IF_DOUBLE(case VLDT_Double : DISPATCH(deviceType, double, VLDT_Double) ; break ;)\
default: assert(false) ; return VLE_Unknown ; \
}

#define DISPATCHCUDNN(dataType) \
error = vl::impl::cov_pool<deviceType, type, dataType>::forward \
(context, \
(type*)output.getMemory(), \
(type const*)data.getMemory(), \
data.getHeight(),data.getWidth(),data.getDepth(),data.getSize()) ;

#define DISPATCHCUDNN2() \
switch (dataType) { \
case VLDT_Float : DISPATCHCUDNN(VLDT_Float) ; break ; \
IF_DOUBLE(case VLDT_Double : DISPATCHCUDNN(VLDT_Double) ; break ;) \
default: assert(false) ; return VLE_Unknown ; \
}

vl::ErrorCode
vl::nncov_pool_forward(vl::Context& context,
                       vl::Tensor output,
                       vl::Tensor data)
{
  vl::ErrorCode error = VLE_Success ;
  vl::DataType dataType = output.getDataType() ;

  switch (output.getDeviceType()) {
    default:
      assert(false) ;
      error = vl::VLE_Unknown ;
      break ;

    case vl::VLDT_CPU:
      DISPATCH2(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
    case vl::VLDT_GPU:
#if ENABLE_CUDNN
//      if (context.getCudaHelper().getCudnnEnabled()) {
//        DISPATCHCUDNN2() ;
//        if (error == vl::VLE_Success) { return error ; }
//        if (error != vl::VLE_Unsupported) { goto done ; }
        /* this case was not supported by CUDNN -- fallback */
//      }
#endif
      DISPATCH2(vl::VLDT_GPU) ;
      break ;
#endif
  }
#if ENABLE_CUDNN
//done:
#endif
  return context.passError(error,__func__) ;
}


#undef DISPATCH
#define DISPATCH(deviceType, type, dataType) \
error = vl::impl::cov_pool<deviceType, type, dataType>::backward \
(context, \
 (type*)derData.getMemory(), \
 (type const*)data.getMemory(),    \
 (type const*)derOutput.getMemory(), \
 derData.getHeight(),derData.getWidth(),derData.getDepth(),derData.getSize()) ;

#undef DISPATCHCUDNN
#define DISPATCHCUDNN(dataType) \
error = vl::impl::cov_pool<deviceType, type, dataType>::backward \
(context, \
 (type*)derData.getMemory(), \
 (type const*)data.getMemory(),    \
 (type const*)derOutput.getMemory(), \
 derData.getHeight(),derData.getWidth(),derData.getDepth(),derData.getSize()) ;

vl::ErrorCode
vl::nncov_pool_backward(vl::Context& context,
                       vl::Tensor derData,
                       vl::Tensor data,
                       vl::Tensor derOutput)
{
  vl::ErrorCode error = vl::VLE_Success ;
  vl::DataType dataType = derOutput.getDataType() ;

  switch (derOutput.getDeviceType()) {
    default:
      assert(false) ;
      error = vl::VLE_Unknown ;
      break ;

    case vl::VLDT_CPU:
      DISPATCH2(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
    case vl::VLDT_GPU:

#if ENABLE_CUDNN
//     if (context.getCudaHelper().getCudnnEnabled()) {
//        DISPATCHCUDNN2() ;
//        if (error == vl::VLE_Success) { return error ; }
//        if (error != vl::VLE_Unsupported) { goto done ; }
        /* this case was not supported by CUDNN -- fallback */
//      }
#endif
      DISPATCH2(vl::VLDT_GPU) ;
      break ;
#endif
  }
#if ENABLE_CUDNN
//done:
#endif
  return context.passError(error,__func__) ;
}

